#include "hip/hip_runtime.h"
#include<iostream>
#include<algorithm>
#include <bits/stdc++.h>
#include<fstream>
#include<random>
#include<cstring>
#include <string.h>
#include<omp.h>
#include <thrust/transform.h>
#include <thrust/transform_reduce.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/iterator/counting_iterator.h>
using namespace std;

// Function to print all sub strings
vector<string> subString(string str, int n) {
    vector<string> str_combinations; 

    // Pick starting point
    for (int len = 1; len <= n; len++) {
        string substring;

        // Pick ending point
        for (int i = 0; i <= n - len; i++) {
            int j = i + len - 1;           
            for (int k = i; k <= j; k++) {
                substring += str[k];
            }

            str_combinations.push_back(substring);
            substring.clear();
        }
    }

    return str_combinations;
}

struct calculoScore
{
    thrust::device_ptr<char> subA;
    char letradeB;
    thrust::device_ptr<int> calc0;

    calculoScore(thrust::device_ptr<char> subA_, char letradeB_, thrust::device_ptr<int> calc0_) :
    subA(subA_),
    letradeB(letradeB_),
    calc0(calc0_) {};
    
    __host__ __device__
    int operator()(const int& x) {

        int score = (letradeB == subA[x-1] ? 2 : -1);

        int diagonal = calc0[x-1] + score;
        int cima = calc0[x] - 1;

        int result;
        if(diagonal >= cima && diagonal > 0) {
            result = diagonal;
        } else if (cima > 0) {
            result = cima;
        } else {
            result = 0;
        }

        return result;
    }
};

struct calculoHorizontal
{
    
    __host__ __device__
    int operator()(const int& i, const int& j) {
        int max, actual, left;

        actual = j;
        left = i - 1;

        if(left >= actual && left > 0) {
            max = left;
        } else if(actual > 0) {
            max = actual;
        } else {
            max = 0;
        }

        return max;
    }
};

int main() {
     // Inicializando as seq. A e B e 
    // seus respectivos tamanhos
    int n, m;
    string a, b;

    cin >> n >> m;
    cin >> a >> b;

    vector<string> a_combinations = subString(a, n);
    vector<string> b_combinations = subString(b, m);
    thrust::counting_iterator<int> c0(1);

    thrust::device_vector<int> maxScores;

    #pragma omp for collapse(2)
    for(uint i = 0; i < a_combinations.size(); i++) {
        for(uint j = 0; j < b_combinations.size(); j++) {

            thrust::device_vector<char> subA(a_combinations[i].begin(), a_combinations[i].end());
            thrust::device_vector<char> subB(b_combinations[j].begin(), b_combinations[j].end()); 

            thrust::device_vector<int> calc[2];
            calc[0].resize(subA.size()+1);
            calc[1].resize(subA.size()+1);
            thrust::fill(calc[0].begin(), calc[0].end(), 0);

            thrust::counting_iterator<int> c1(int(subA.size()) + 1);

            for(int k = 0; k < subB.size(); k++) {
                char letradeB = subB[k];
                thrust::transform(c0, c1, calc[1].begin() + 1, calculoScore(subA.data(), letradeB, calc[0].data()));
                thrust::inclusive_scan(calc[1].begin() + 1, calc[1].end(), calc[0].begin() + 1, calculoHorizontal());
            }


            maxScores.push_back(calc[0].back());
        }    
    }

    cout << thrust::reduce(maxScores.begin(), maxScores.end(), 0, thrust::maximum<int>()) << endl;
    return 0;
}
